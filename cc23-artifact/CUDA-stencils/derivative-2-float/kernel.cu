#include <stdio.h>
#include "hip/hip_runtime.h"
#define max(x,y)  ((x) > (y)? (x) : (y))
#define min(x,y)  ((x) < (y)? (x) : (y))
#define ceil(a,b) ((a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1)

void check_error (const char* message) {
	hipError_t error = hipGetLastError ();
	if (error != hipSuccess) {
		printf ("CUDA error : %s, %s\n", message, hipGetErrorString (error));
		exit(-1);
	}
}

__global__ void curvi (float * __restrict__ r1, float *__restrict__ in_u1, float * __restrict__ in_u2, float *__restrict__ in_u3, float * __restrict__ in_mu, float * __restrict__ in_la, float * __restrict__ in_met1, float * __restrict__ in_met2, float * __restrict__ in_met3, float * __restrict__ in_met4, float * __restrict__ strx, float * __restrict__ stry, float c1, float c2, int N) {
	//Determing the block's indices
	int blockdim_k= (int)(blockDim.x);
	int k0 = (int)(blockIdx.x)*(blockdim_k);
	int k = max (k0, 0) + (int)(threadIdx.x);
	int blockdim_j= (int)(blockDim.y);
	int j0 = (int)(blockIdx.y)*(blockdim_j);
	int j = max (j0, 0) + (int)(threadIdx.y);

	float (*u1)[304][304] = (float (*)[304][304])in_u1;
	float (*u2)[304][304] = (float (*)[304][304])in_u2;
	float (*u3)[304][304] = (float (*)[304][304])in_u3;
	float (*mu)[304][304] = (float (*)[304][304])in_mu;
	float (*la)[304][304] = (float (*)[304][304])in_la;
	float (*met1)[304][304] = (float (*)[304][304])in_met1;
	float (*met2)[304][304] = (float (*)[304][304])in_met2;
	float (*met3)[304][304] = (float (*)[304][304])in_met3;
	float (*met4)[304][304] = (float (*)[304][304])in_met4;

	if (j>=2 & k>=2 & j<=N-3 & k<=N-3) {
		for (int i=2; i<=N-3; i++) {
		r1[i*N*N+j*N+k] +=
			c2*(  mu[i][j+2][k]*met1[i][j+2][k]*met1[i][j+2][k]*(
						c2*(u2[i+2][j+2][k]-u2[i-2][j+2][k]) +
						c1*(u2[i+1][j+2][k]-u2[i-1][j+2][k])    )
					+  mu[i][j-2][k]*met1[i][j-2][k]*met1[i][j-2][k]*(
						c2*(u2[i+2][j-2][k]-u2[i-2][j-2][k])+
						c1*(u2[i+1][j-2][k]-u2[i-1][j-2][k])     )
			   ) +
			c1*(  mu[i][j+1][k]*met1[i][j+1][k]*met1[i][j+1][k]*(
						c2*(u2[i+2][j+1][k]-u2[i-2][j+1][k]) +
						c1*(u2[i+1][j+1][k]-u2[i-1][j+1][k])  )
					+ mu[i][j-1][k]*met1[i][j-1][k]*met1[i][j-1][k]*(
						c2*(u2[i+2][j-1][k]-u2[i-2][j-1][k]) +
						c1*(u2[i+1][j-1][k]-u2[i-1][j-1][k])))
			+
			c2*(  la[i+2][j][k]*met1[i+2][j][k]*met1[i+2][j][k]*(
						c2*(u2[i+2][j+2][k]-u2[i+2][j-2][k]) +
						c1*(u2[i+2][j+1][k]-u2[i+2][j-1][k])    )
					+ la[i-2][j][k]*met1[i-2][j][k]*met1[i-2][j][k]*(
						c2*(u2[i-2][j+2][k]-u2[i-2][j-2][k])+
						c1*(u2[i-2][j+1][k]-u2[i-2][j-1][k])     )
			   ) +
			c1*(  la[i+1][j][k]*met1[i+1][j][k]*met1[i+1][j][k]*(
						c2*(u2[i+1][j+2][k]-u2[i+1][j-2][k]) +
						c1*(u2[i+1][j+1][k]-u2[i+1][j-1][k])  )
					+ la[i-1][j][k]*met1[i-1][j][k]*met1[i-1][j][k]*(
						c2*(u2[i-1][j+2][k]-u2[i-1][j-2][k]) +
						c1*(u2[i-1][j+1][k]-u2[i-1][j-1][k])));

		r1[i*N*N+j*N+k] += c2*(
				(2*mu[i][j][k+2]+la[i][j][k+2])*met2[i][j][k+2]*met1[i][j][k+2]*(
					c2*(u1[i+2][j][k+2]-u1[i-2][j][k+2]) +
					c1*(u1[i+1][j][k+2]-u1[i-1][j][k+2])   )*strx[i]*stry[j]
				+ mu[i][j][k+2]*met3[i][j][k+2]*met1[i][j][k+2]*(
					c2*(u2[i+2][j][k+2]-u2[i-2][j][k+2]) +
					c1*(u2[i+1][j][k+2]-u2[i-1][j][k+2])  )
				+ mu[i][j][k+2]*met4[i][j][k+2]*met1[i][j][k+2]*(
					c2*(u3[i+2][j][k+2]-u3[i-2][j][k+2]) +
					c1*(u3[i+1][j][k+2]-u3[i-1][j][k+2])  )*stry[j]
				+ ((2*mu[i][j][k-2]+la[i][j][k-2])*met2[i][j][k-2]*met1[i][j][k-2]*(
						c2*(u1[i+2][j][k-2]-u1[i-2][j][k-2]) +
						c1*(u1[i+1][j][k-2]-u1[i-1][j][k-2])  )*strx[i]*stry[j]
					+ mu[i][j][k-2]*met3[i][j][k-2]*met1[i][j][k-2]*(
						c2*(u2[i+2][j][k-2]-u2[i-2][j][k-2]) +
						c1*(u2[i+1][j][k-2]-u2[i-1][j][k-2])   )
					+ mu[i][j][k-2]*met4[i][j][k-2]*met1[i][j][k-2]*(
						c2*(u3[i+2][j][k-2]-u3[i-2][j][k-2]) +
						c1*(u3[i+1][j][k-2]-u3[i-1][j][k-2])   )*stry[j] )
				) + c1*(
					(2*mu[i][j][k+1]+la[i][j][k+1])*met2[i][j][k+1]*met1[i][j][k+1]*(
						c2*(u1[i+2][j][k+1]-u1[i-2][j][k+1]) +
						c1*(u1[i+1][j][k+1]-u1[i-1][j][k+1]) )*strx[i+2]*stry[j]
					+ mu[i][j][k+1]*met3[i][j][k+1]*met1[i][j][k+1]*(
						c2*(u2[i+2][j][k+1]-u2[i-2][j][k+1]) +
						c1*(u2[i+1][j][k+1]-u2[i-1][j][k+1]) )
					+ mu[i][j][k+1]*met4[i][j][k+1]*met1[i][j][k+1]*(
						c2*(u3[i+2][j][k+1]-u3[i-2][j][k+1]) +
						c1*(u3[i+1][j][k+1]-u3[i-1][j][k+1])  )*stry[j]
					+ ((2*mu[i][j][k-1]+la[i][j][k-1])*met2[i][j][k-1]*met1[i][j][k-1]*(
							c2*(u1[i+2][j][k-1]-u1[i-2][j][k-1]) +
							c1*(u1[i+1][j][k-1]-u1[i-1][j][k-1]) )*strx[i-2]*stry[j]
						+ mu[i][j][k-1]*met3[i][j][k-1]*met1[i][j][k-1]*(
							c2*(u2[i+2][j][k-1]-u2[i-2][j][k-1]) +
							c1*(u2[i+1][j][k-1]-u2[i-1][j][k-1]) )
						+ mu[i][j][k-1]*met4[i][j][k-1]*met1[i][j][k-1]*(
							c2*(u3[i+2][j][k-1]-u3[i-2][j][k-1]) +
							c1*(u3[i+1][j][k-1]-u3[i-1][j][k-1])   )*stry[j]  ) );

		r1[i*N*N+j*N+k] += ( c2*(
					(2*mu[i+2][j][k]+la[i+2][j][k])*met2[i+2][j][k]*met1[i+2][j][k]*(
						c2*(u1[i+2][j][k+2]-u1[i+2][j][k-2]) +
						c1*(u1[i+2][j][k+1]-u1[i+2][j][k-1])   )*strx[i]
					+ la[i+2][j][k]*met3[i+2][j][k]*met1[i+2][j][k]*(
						c2*(u2[i+2][j][k+2]-u2[i+2][j][k-2]) +
						c1*(u2[i+2][j][k+1]-u2[i+2][j][k-1])  )*stry[j]
					+ la[i+2][j][k]*met4[i+2][j][k]*met1[i+2][j][k]*(
						c2*(u3[i+2][j][k+2]-u3[i+2][j][k-2]) +
						c1*(u3[i+2][j][k+1]-u3[i+2][j][k-1])  )
					+ ((2*mu[i-2][j][k]+la[i-2][j][k])*met2[i-2][j][k]*met1[i-2][j][k]*(
							c2*(u1[i-2][j][k+2]-u1[i-2][j][k-2]) +
							c1*(u1[i-2][j][k+1]-u1[i-2][j][k-1])  )*strx[i]
						+ la[i-2][j][k]*met3[i-2][j][k]*met1[i-2][j][k]*(
							c2*(u2[i-2][j][k+2]-u2[i-2][j][k-2]) +
							c1*(u2[i-2][j][k+1]-u2[i-2][j][k-1])   )*stry[j]
						+ la[i-2][j][k]*met4[i-2][j][k]*met1[i-2][j][k]*(
							c2*(u3[i-2][j][k+2]-u3[i-2][j][k-2]) +
							c1*(u3[i-2][j][k+1]-u3[i-2][j][k-1])   ) )
				    ) + c1*(
					    (2*mu[i+1][j][k]+la[i+1][j][k])*met2[i+1][j][k]*met1[i+1][j][k]*(
						    c2*(u1[i+1][j][k+2]-u1[i+1][j][k-2]) +
						    c1*(u1[i+1][j][k+1]-u1[i+1][j][k-1]) )*strx[i]
					    + la[i+1][j][k]*met3[i+1][j][k]*met1[i+1][j][k]*(
						    c2*(u2[i+1][j][k+2]-u2[i+1][j][k-2]) +
						    c1*(u2[i+1][j][k+1]-u2[i+1][j][k-1]) )*stry[j]
					    + la[i+1][j][k]*met4[i+1][j][k]*met1[i+1][j][k]*(
						    c2*(u3[i+1][j][k+2]-u3[i+1][j][k-2]) +
						    c1*(u3[i+1][j][k+1]-u3[i+1][j][k-1])  )
					    + ((2*mu[i-1][j][k]+la[i-1][j][k])*met2[i-1][j][k]*met1[i-1][j][k]*(
							    c2*(u1[i-1][j][k+2]-u1[i-1][j][k-2]) +
							    c1*(u1[i-1][j][k+1]-u1[i-1][j][k-1]) )*strx[i]
						    + la[i-1][j][k]*met3[i-1][j][k]*met1[i-1][j][k]*(
							    c2*(u2[i-1][j][k+2]-u2[i-1][j][k-2]) +
							    c1*(u2[i-1][j][k+1]-u2[i-1][j][k-1]) )*stry[j]
						    + la[i-1][j][k]*met4[i-1][j][k]*met1[i-1][j][k]*(
							    c2*(u3[i-1][j][k+2]-u3[i-1][j][k-2]) +
							    c1*(u3[i-1][j][k+1]-u3[i-1][j][k-1])   )  ) ) )*stry[j];

		r1[i*N*N+j*N+k] += c2*(
				mu[i][j+2][k]*met3[i][j+2][k]*met1[i][j+2][k]*(
					c2*(u1[i][j+2][k+2]-u1[i][j+2][k-2]) +
					c1*(u1[i][j+2][k+1]-u1[i][j+2][k-1])   )*stry[j+1]*strx[i]
				+ mu[i][j+2][k]*met2[i][j+2][k]*met1[i][j+2][k]*(
					c2*(u2[i][j+2][k+2]-u2[i][j+2][k-2]) +
					c1*(u2[i][j+2][k+1]-u2[i][j+2][k-1])  )
				+ ( mu[i][j-2][k]*met3[i][j-2][k]*met1[i][j-2][k]*(
						c2*(u1[i][j-2][k+2]-u1[i][j-2][k-2]) +
						c1*(u1[i][j-2][k+1]-u1[i][j-2][k-1])  )*stry[j]*strx[i]
					+ mu[i][j-2][k]*met2[i][j-2][k]*met1[i][j-2][k]*(
						c2*(u2[i][j-2][k+2]-u2[i][j-2][k-2]) +
						c1*(u2[i][j-2][k+1]-u2[i][j-2][k-1])   ) )
				) + c1*(
					mu[i][j+1][k]*met3[i][j+1][k]*met1[i][j+1][k]*(
						c2*(u1[i][j+1][k+2]-u1[i][j+1][k-2]) +
						c1*(u1[i][j+1][k+1]-u1[i][j+1][k-1]) )*stry[j-1]*strx[i]
					+ mu[i][j+1][k]*met2[i][j+1][k]*met1[i][j+1][k]*(
						c2*(u2[i][j+1][k+2]-u2[i][j+1][k-2]) +
						c1*(u2[i][j+1][k+1]-u2[i][j+1][k-1]) )
					+ ( mu[i][j-1][k]*met3[i][j-1][k]*met1[i][j-1][k]*(
							c2*(u1[i][j-1][k+2]-u1[i][j-1][k-2]) +
							c1*(u1[i][j-1][k+1]-u1[i][j-1][k-1]) )*stry[j]*strx[i]
						+ mu[i][j-1][k]*met2[i][j-1][k]*met1[i][j-1][k]*(
							c2*(u2[i][j-1][k+2]-u2[i][j-1][k-2]) +
							c1*(u2[i][j-1][k+1]-u2[i][j-1][k-1]) ) ) );


		r1[i*N*N+j*N+k] += c2*(
				mu[i][j][k+2]*met3[i][j][k+2]*met1[i][j][k+2]*(
					c2*(u1[i][j+2][k+2]-u1[i][j-2][k+2]) +
					c1*(u1[i][j+1][k+2]-u1[i][j-1][k+2])   )*stry[j+2]*strx[i]
				+ la[i][j][k+2]*met2[i][j][k+2]*met1[i][j][k+2]*(
					c2*(u2[i][j+2][k+2]-u2[i][j-2][k+2]) +
					c1*(u2[i][j+1][k+2]-u2[i][j-1][k+2])  )
				+ ( mu[i][j][k-2]*met3[i][j][k-2]*met1[i][j][k-2]*(
						c2*(u1[i][j+2][k-2]-u1[i][j-2][k-2]) +
						c1*(u1[i][j+1][k-2]-u1[i][j-1][k-2])  )*stry[j]*strx[i]
					+ la[i][j][k-2]*met2[i][j][k-2]*met1[i][j][k-2]*(
						c2*(u2[i][j+2][k-2]-u2[i][j-2][k-2]) +
						c1*(u2[i][j+1][k-2]-u2[i][j-1][k-2])   ) )
				) + c1*(
					mu[i][j][k+1]*met3[i][j][k+1]*met1[i][j][k+1]*(
						c2*(u1[i][j+2][k+1]-u1[i][j-2][k+1]) +
						c1*(u1[i][j+1][k+1]-u1[i][j-1][k+1]) )*stry[j-2]*strx[i]
					+ la[i][j][k+1]*met2[i][j][k+1]*met1[i][j][k+1]*(
						c2*(u2[i][j+2][k+1]-u2[i][j-2][k+1]) +
						c1*(u2[i][j+1][k+1]-u2[i][j-1][k+1]) )
					+ ( mu[i][j][k-1]*met3[i][j][k-1]*met1[i][j][k-1]*(
							c2*(u1[i][j+2][k-1]-u1[i][j-2][k-1]) +
							c1*(u1[i][j+1][k-1]-u1[i][j-1][k-1]) )*stry[j]*strx[i]
						+ la[i][j][k-1]*met2[i][j][k-1]*met1[i][j][k-1]*(
							c2*(u2[i][j+2][k-1]-u2[i][j-2][k-1]) +
							c1*(u2[i][j+1][k-1]-u2[i][j-1][k-1]) ) ) );

		}
	} 
}

extern "C" void host_code (float *h_r1, float *h_u1, float *h_u2, float *h_u3,  float *h_mu, float *h_la, float *h_met1, float *h_met2, float *h_met3, float *h_met4, float *h_strx, float *h_stry, float c1, float c2, int N) {
	float *r1;
	hipMalloc (&r1, sizeof(float)*N*N*N);
	check_error ("Failed to allocate device memory for r1\n");
	hipMemcpy (r1, h_r1, sizeof(float)*N*N*N, hipMemcpyHostToDevice);
	float *u1;
	hipMalloc (&u1, sizeof(float)*N*N*N);
	check_error ("Failed to allocate device memory for u1\n");
	hipMemcpy (u1, h_u1, sizeof(float)*N*N*N, hipMemcpyHostToDevice);
	float *u2;
	hipMalloc (&u2, sizeof(float)*N*N*N);
	check_error ("Failed to allocate device memory for u2\n");
	hipMemcpy (u2, h_u2, sizeof(float)*N*N*N, hipMemcpyHostToDevice);
	float *u3;
	hipMalloc (&u3, sizeof(float)*N*N*N);
	check_error ("Failed to allocate device memory for u3\n");
	hipMemcpy (u3, h_u3, sizeof(float)*N*N*N, hipMemcpyHostToDevice);
	float *mu;
	hipMalloc (&mu, sizeof(float)*N*N*N);
	check_error ("Failed to allocate device memory for mu\n");
	hipMemcpy (mu, h_mu, sizeof(float)*N*N*N, hipMemcpyHostToDevice);
	float *la;
	hipMalloc (&la, sizeof(float)*N*N*N);
	check_error ("Failed to allocate device memory for la\n");
	hipMemcpy (la, h_la, sizeof(float)*N*N*N, hipMemcpyHostToDevice);
	float *met1;
	hipMalloc (&met1, sizeof(float)*N*N*N);
	check_error ("Failed to allocate device memory for met1\n");
	hipMemcpy (met1, h_met1, sizeof(float)*N*N*N, hipMemcpyHostToDevice);
	float *met2;
	hipMalloc (&met2, sizeof(float)*N*N*N);
	check_error ("Failed to allocate device memory for met2\n");
	hipMemcpy (met2, h_met2, sizeof(float)*N*N*N, hipMemcpyHostToDevice);
	float *met3;
	hipMalloc (&met3, sizeof(float)*N*N*N);
	check_error ("Failed to allocate device memory for met3\n");
	hipMemcpy (met3, h_met3, sizeof(float)*N*N*N, hipMemcpyHostToDevice);
	float *met4;
	hipMalloc (&met4, sizeof(float)*N*N*N);
	check_error ("Failed to allocate device memory for met4\n");
	hipMemcpy (met4, h_met4, sizeof(float)*N*N*N, hipMemcpyHostToDevice);
	float *strx;
	hipMalloc (&strx, sizeof(float)*N);
	check_error ("Failed to allocate device memory for strx\n");
	hipMemcpy (strx, h_strx, sizeof(float)*N, hipMemcpyHostToDevice);
	float *stry;
	hipMalloc (&stry, sizeof(float)*N);
	check_error ("Failed to allocate device memory for stry\n");
	hipMemcpy (stry, h_stry, sizeof(float)*N, hipMemcpyHostToDevice);

	dim3 blockconfig (32, 8);
	dim3 gridconfig (ceil(N, blockconfig.x), ceil(N, blockconfig.y), 1);

	curvi <<<gridconfig, blockconfig>>> (r1, u1, u2, u3, mu, la, met1, met2, met3, met4, strx, stry, c1, c2, N);
	hipMemcpy (h_r1, r1, sizeof(float)*N*N*N, hipMemcpyDeviceToHost);
}
