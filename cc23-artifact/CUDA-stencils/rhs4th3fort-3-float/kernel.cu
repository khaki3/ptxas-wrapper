#include <stdio.h>
#include "hip/hip_runtime.h"
#define max(x,y)  ((x) > (y)? (x) : (y))
#define min(x,y)  ((x) < (y)? (x) : (y))
#define ceil(a,b) ((a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1)

void check_error (const char* message) {
	hipError_t error = hipGetLastError ();
	if (error != hipSuccess) {
		printf ("CUDA error : %s, %s\n", message, hipGetErrorString (error));
		exit(-1);
	}
}

__global__ void sw4 (float * __restrict__ uacc_0, float * __restrict__ uacc_1, float * __restrict__ uacc_2, float * __restrict__ u_0, float * __restrict__ u_1, float * __restrict__ u_2, float * __restrict__ mu, float * __restrict__ la, float * __restrict__ strx, float * __restrict__ stry, float * __restrict__ strz, int N) {
	//Determing the block's indices
	int blockdim_i= (int)(blockDim.x);
	int i0 = (int)(blockIdx.x)*(blockdim_i);
	int i = max (i0, 0) + (int)(threadIdx.x);
	int blockdim_j= (int)(blockDim.y);
	int j0 = (int)(blockIdx.y)*(blockdim_j);
	int j = max (j0, 0) + (int)(threadIdx.y);
	int blockdim_k= (int)(blockDim.z);
	int k0 = (int)(blockIdx.z)*(blockdim_k);
	int k = max (k0, 0) + (int)(threadIdx.z);

	// Assumptions 
	int a1 = 1;
	float h = 3.7;
	float cof = 1e0 / ( h *  h);

	if (i>=2 & j>=2 & k>=2 & i<=N-3 & j<=N-3 & k<=N-3) {
			/* 28 * 3 = 84 flops */
			float mux1 = mu[k*N*N+j*N+i-1] * strx[i-1] - 3e0 / 4 * (mu[k*N*N+j*N+i] * strx[i] + mu[k*N*N+j*N+i-2] * strx[i-2]);
			float mux2 = mu[k*N*N+j*N+i-2] * strx[i-2] + mu[k*N*N+j*N+i+1] * strx[i+1] + 3 * (mu[k*N*N+j*N+i] * strx[i] + mu[k*N*N+j*N+i-1] * strx[i-1]);
			float mux3 = mu[k*N*N+j*N+i-1] * strx[i-1] + mu[k*N*N+j*N+i+2] * strx[i+2] + 3 * (mu[k*N*N+j*N+i+1] * strx[i+1] + mu[k*N*N+j*N+i] * strx[i]);
			float mux4 = mu[k*N*N+j*N+i+1] * strx[i+1] - 3e0 / 4 * (mu[k*N*N+j*N+i] * strx[i] + mu[k*N*N+j*N+i+2] * strx[i+2]);
			float muy1 = mu[k*N*N+(j-1)*N+i] * stry[j-1] - 3e0 / 4 * (mu[k*N*N+j*N+i] * stry[j] + mu[k*N*N+(j-2)*N+i] * stry[j-2]);
			float muy2 = mu[k*N*N+(j-2)*N+i] * stry[j-2] + mu[k*N*N+(j+1)*N+i] * stry[j+1] + 3 * (mu[k*N*N+j*N+i] * stry[j] + mu[k*N*N+(j-1)*N+i] * stry[j-1]);
			float muy3 = mu[k*N*N+(j-1)*N+i] * stry[j-1] + mu[k*N*N+(j+2)*N+i] * stry[j+2] + 3 * (mu[k*N*N+(j+1)*N+i] * stry[j+1] + mu[k*N*N+j*N+i] * stry[j]);
			float muy4 = mu[k*N*N+(j+1)*N+i] * stry[j+1] - 3e0 / 4 * (mu[k*N*N+j*N+i] * stry[j] + mu[k*N*N+(j+2)*N+i] * stry[j+2]);
			float muz1 = mu[(k-1)*N*N+j*N+i] * strz[k-1] - 3e0 / 4 * (mu[k*N*N+j*N+i] * strz[k] + mu[(k-2)*N*N+j*N+i] * strz[k-2]);
			float muz2 = mu[(k-2)*N*N+j*N+i] * strz[k-2] + mu[(k+1)*N*N+j*N+i] * strz[k+1] + 3 * (mu[k*N*N+j*N+i] * strz[k] + mu[(k-1)*N*N+j*N+i] * strz[k-1]);
			float muz3 = mu[(k-1)*N*N+j*N+i] * strz[k-1] + mu[(k+2)*N*N+j*N+i] * strz[k+2] + 3 * (mu[(k+1)*N*N+j*N+i] * strz[k+1] + mu[k*N*N+j*N+i] * strz[k]);
			float muz4 = mu[(k+1)*N*N+j*N+i] * strz[k+1] - 3e0 / 4 * (mu[k*N*N+j*N+i] * strz[k] + mu[(k+2)*N*N+j*N+i] * strz[k+2]);

			/* 78 * 3 = 234 flops */
			float r1 = 1e0 / 6 * (strx[i] * ((2 * mux1 + la[k*N*N+j*N+i-1] * strx[i-1] - 3e0 / 4 * (la[k*N*N+j*N+i] * strx[i] + la[k*N*N+j*N+i-2] * strx[i-2])) * (u_0[k*N*N+j*N+i-2] - u_0[k*N*N+j*N+i]) + (2 * mux2 + la[k*N*N+j*N+i-2] * strx[i-2] + la[k*N*N+j*N+i+1] * strx[i+1] + 3 * (la[k*N*N+j*N+i] * strx[i] + la[k*N*N+j*N+i-1] * strx[i-1])) * (u_0[k*N*N+j*N+i-1] - u_0[k*N*N+j*N+i]) + (2 * mux3 + la[k*N*N+j*N+i-1] * strx[i-1] + la[k*N*N+j*N+i+2] * strx[i+2] + 3 * (la[k*N*N+j*N+i+1] * strx[i+1] + la[k*N*N+j*N+i] * strx[i])) * (u_0[k*N*N+j*N+i+1] - u_0[k*N*N+j*N+i]) + (2 * mux4 + la[k*N*N+j*N+i+1] * strx[i+1] - 3e0 / 4 * (la[k*N*N+j*N+i] * strx[i] + la[k*N*N+j*N+i+2] * strx[i+2])) * (u_0[k*N*N+j*N+i+2] - u_0[k*N*N+j*N+i])) + stry[j] * (muy1 * (u_0[k*N*N+(j-2)*N+i] - u_0[k*N*N+j*N+i]) + muy2 * (u_0[k*N*N+(j-1)*N+i] - u_0[k*N*N+j*N+i]) + muy3 * (u_0[k*N*N+(j+1)*N+i] - u_0[k*N*N+j*N+i]) + muy4 * (u_0[k*N*N+(j+2)*N+i] - u_0[k*N*N+j*N+i])) + strz[k] * (muz1 * (u_0[(k-2)*N*N+j*N+i] - u_0[k*N*N+j*N+i]) + muz2 * (u_0[(k-1)*N*N+j*N+i] - u_0[k*N*N+j*N+i]) + muz3 * (u_0[(k+1)*N*N+j*N+i] - u_0[k*N*N+j*N+i]) + muz4 * (u_0[(k+2)*N*N+j*N+i] - u_0[k*N*N+j*N+i])));
			float r2 = 1e0 / 6 * (strx[i] * (mux1 * (u_1[k*N*N+j*N+i-2] - u_1[k*N*N+j*N+i]) + mux2 * (u_1[k*N*N+j*N+i-1] - u_1[k*N*N+j*N+i]) + mux3 * (u_1[k*N*N+j*N+i+1] - u_1[k*N*N+j*N+i]) + mux4 * (u_1[k*N*N+j*N+i+2] - u_1[k*N*N+j*N+i])) + stry[j] * ((2 * muy1 + la[k*N*N+(j-1)*N+i] * stry[j-1] - 3e0 / 4 * (la[k*N*N+j*N+i] * stry[j] + la[k*N*N+(j-2)*N+i] * stry[j-2])) * (u_1[k*N*N+(j-2)*N+i] - u_1[k*N*N+j*N+i]) + (2 * muy2 + la[k*N*N+(j-2)*N+i] * stry[j-2] + la[k*N*N+(j+1)*N+i] * stry[j+1] + 3 * (la[k*N*N+j*N+i] * stry[j] + la[k*N*N+(j-1)*N+i] * stry[j-1])) * (u_1[k*N*N+(j-1)*N+i] - u_1[k*N*N+j*N+i]) + (2 * muy3 + la[k*N*N+(j-1)*N+i] * stry[j-1] + la[k*N*N+(j+2)*N+i] * stry[j+2] + 3 * (la[k*N*N+(j+1)*N+i] * stry[j+1] + la[k*N*N+j*N+i] * stry[j])) * (u_1[k*N*N+(j+1)*N+i] - u_1[k*N*N+j*N+i]) + (2 * muy4 + la[k*N*N+(j+1)*N+i] * stry[j+1] - 3e0 / 4 * (la[k*N*N+j*N+i] * stry[j] + la[k*N*N+(j+2)*N+i] * stry[j+2])) * (u_1[k*N*N+(j+2)*N+i] - u_1[k*N*N+j*N+i])) + strz[k] * (muz1 * (u_1[(k-2)*N*N+j*N+i] - u_1[k*N*N+j*N+i]) + muz2 * (u_1[(k-1)*N*N+j*N+i] - u_1[k*N*N+j*N+i]) + muz3 * (u_1[(k+1)*N*N+j*N+i] - u_1[k*N*N+j*N+i]) + muz4 * (u_1[(k+2)*N*N+j*N+i] - u_1[k*N*N+j*N+i])));
			float r3 = 1e0 / 6 * (strx[i] * (mux1 * (u_2[k*N*N+j*N+i-2] - u_2[k*N*N+j*N+i]) + mux2 * (u_2[k*N*N+j*N+i-1] - u_2[k*N*N+j*N+i]) + mux3 * (u_2[k*N*N+j*N+i+1] - u_2[k*N*N+j*N+i]) + mux4 * (u_2[k*N*N+j*N+i+2] - u_2[k*N*N+j*N+i])) + stry[j] * (muy1 * (u_2[k*N*N+(j-2)*N+i] - u_2[k*N*N+j*N+i]) + muy2 * (u_2[k*N*N+(j-1)*N+i] - u_2[k*N*N+j*N+i]) + muy3 * (u_2[k*N*N+(j+1)*N+i] - u_2[k*N*N+j*N+i]) + muy4 * (u_2[k*N*N+(j+2)*N+i] - u_2[k*N*N+j*N+i])) + strz[k] * ((2 * muz1 + la[(k-1)*N*N+j*N+i] * strz[k-1] - 3e0 / 4 * (la[k*N*N+j*N+i] * strz[k] + la[(k-2)*N*N+j*N+i] * strz[k-2])) * (u_2[(k-2)*N*N+j*N+i] - u_2[k*N*N+j*N+i]) + (2 * muz2 + la[(k-2)*N*N+j*N+i] * strz[k-2] + la[(k+1)*N*N+j*N+i] * strz[k+1] + 3 * (la[k*N*N+j*N+i] * strz[k] + la[(k-1)*N*N+j*N+i] * strz[k-1])) * (u_2[(k-1)*N*N+j*N+i] - u_2[k*N*N+j*N+i]) + (2 * muz3 + la[(k-1)*N*N+j*N+i] * strz[k-1] + la[(k+2)*N*N+j*N+i] * strz[k+2] + 3 * (la[(k+1)*N*N+j*N+i] * strz[k+1] + la[k*N*N+j*N+i] * strz[k])) * (u_2[(k+1)*N*N+j*N+i] - u_2[k*N*N+j*N+i]) + (2 * muz4 + la[(k+1)*N*N+j*N+i] * strz[k+1] - 3e0 / 4 * (la[k*N*N+j*N+i] * strz[k] + la[(k+2)*N*N+j*N+i] * strz[k+2])) * (u_2[(k+2)*N*N+j*N+i] - u_2[k*N*N+j*N+i])));

			/* 120 * 3 = 360 flops */
			r1 = r1 + strx[i] * stry[j] * (1e0 / 144) * (la[k*N*N+j*N+i-2] * (u_1[k*N*N+(j-2)*N+i-2] - u_1[k*N*N+(j+2)*N+i-2] + 8 * (-u_1[k*N*N+(j-1)*N+i-2] + u_1[k*N*N+(j+1)*N+i-2])) - 8 * (la[k*N*N+j*N+i-1] * (u_1[k*N*N+(j-2)*N+i-1] - u_1[k*N*N+(j+2)*N+i-1] + 8 * (-u_1[k*N*N+(j-1)*N+i-1] + u_1[k*N*N+(j+1)*N+i-1]))) + 8 * (la[k*N*N+j*N+i+1] * (u_1[k*N*N+(j-2)*N+i+1] - u_1[k*N*N+(j+2)*N+i+1] + 8 * (-u_1[k*N*N+(j-1)*N+i+1] + u_1[k*N*N+(j+1)*N+i+1]))) - (la[k*N*N+j*N+i+2] * (u_1[k*N*N+(j-2)*N+i+2] - u_1[k*N*N+(j+2)*N+i+2] + 8 * (-u_1[k*N*N+(j-1)*N+i+2] + u_1[k*N*N+(j+1)*N+i+2])))) + strx[i] * strz[k] * (1e0 / 144) * (la[k*N*N+j*N+i-2] * (u_2[(k-2)*N*N+j*N+i-2] - u_2[(k+2)*N*N+j*N+i-2] + 8 * (-u_2[(k-1)*N*N+j*N+i-2] + u_2[(k+1)*N*N+j*N+i-2])) - 8 * (la[k*N*N+j*N+i-1] * (u_2[(k-2)*N*N+j*N+i-1] - u_2[(k+2)*N*N+j*N+i-1] + 8 * (-u_2[(k-1)*N*N+j*N+i-1] + u_2[(k+1)*N*N+j*N+i-1]))) + 8 * (la[k*N*N+j*N+i+1] * (u_2[(k-2)*N*N+j*N+i+1] - u_2[(k+2)*N*N+j*N+i+1] + 8 * (-u_2[(k-1)*N*N+j*N+i+1] + u_2[(k+1)*N*N+j*N+i+1]))) - (la[k*N*N+j*N+i+2] * (u_2[(k-2)*N*N+j*N+i+2] - u_2[(k+2)*N*N+j*N+i+2] + 8 * (-u_2[(k-1)*N*N+j*N+i+2] + u_2[(k+1)*N*N+j*N+i+2])))) + strx[i] * stry[j] * (1e0 / 144) * (mu[k*N*N+(j-2)*N+i] * (u_1[k*N*N+(j-2)*N+i-2] - u_1[k*N*N+(j-2)*N+i+2] + 8 * (-u_1[k*N*N+(j-2)*N+i-1] + u_1[k*N*N+(j-2)*N+i+1])) - 8 * (mu[k*N*N+(j-1)*N+i] * (u_1[k*N*N+(j-1)*N+i-2] - u_1[k*N*N+(j-1)*N+i+2] + 8 * (-u_1[k*N*N+(j-1)*N+i-1] + u_1[k*N*N+(j-1)*N+i+1]))) + 8 * (mu[k*N*N+(j+1)*N+i] * (u_1[k*N*N+(j+1)*N+i-2] - u_1[k*N*N+(j+1)*N+i+2] + 8 * (-u_1[k*N*N+(j+1)*N+i-1] + u_1[k*N*N+(j+1)*N+i+1]))) - (mu[k*N*N+(j+2)*N+i] * (u_1[k*N*N+(j+2)*N+i-2] - u_1[k*N*N+(j+2)*N+i+2] + 8 * (-u_1[k*N*N+(j+2)*N+i-1] + u_1[k*N*N+(j+2)*N+i+1])))) + strx[i] * strz[k] * (1e0 / 144) * (mu[(k-2)*N*N+j*N+i] * (u_2[(k-2)*N*N+j*N+i-2] - u_2[(k-2)*N*N+j*N+i+2] + 8 * (-u_2[(k-2)*N*N+j*N+i-1] + u_2[(k-2)*N*N+j*N+i+1])) - 8 * (mu[(k-1)*N*N+j*N+i] * (u_2[(k-1)*N*N+j*N+i-2] - u_2[(k-1)*N*N+j*N+i+2] + 8 * (-u_2[(k-1)*N*N+j*N+i-1] + u_2[(k-1)*N*N+j*N+i+1]))) + 8 * (mu[(k+1)*N*N+j*N+i] * (u_2[(k+1)*N*N+j*N+i-2] - u_2[(k+1)*N*N+j*N+i+2] + 8 * (-u_2[(k+1)*N*N+j*N+i-1] + u_2[(k+1)*N*N+j*N+i+1]))) - (mu[(k+2)*N*N+j*N+i] * (u_2[(k+2)*N*N+j*N+i-2] - u_2[(k+2)*N*N+j*N+i+2] + 8 * (-u_2[(k+2)*N*N+j*N+i-1] + u_2[(k+2)*N*N+j*N+i+1])))); 
			r2 = r2 + strx[i] * stry[j] * (1e0 / 144) * (mu[k*N*N+j*N+i-2] * (u_0[k*N*N+(j-2)*N+i-2] - u_0[k*N*N+(j+2)*N+i-2] + 8 * (-u_0[k*N*N+(j-1)*N+i-2] + u_0[k*N*N+(j+1)*N+i-2])) - 8 * (mu[k*N*N+j*N+i-1] * (u_0[k*N*N+(j-2)*N+i-1] - u_0[k*N*N+(j+2)*N+i-1] + 8 * (-u_0[k*N*N+(j-1)*N+i-1] + u_0[k*N*N+(j+1)*N+i-1]))) + 8 * (mu[k*N*N+j*N+i+1] * (u_0[k*N*N+(j-2)*N+i+1] - u_0[k*N*N+(j+2)*N+i+1] + 8 * (-u_0[k*N*N+(j-1)*N+i+1] + u_0[k*N*N+(j+1)*N+i+1]))) - (mu[k*N*N+j*N+i+2] * (u_0[k*N*N+(j-2)*N+i+2] - u_0[k*N*N+(j+2)*N+i+2] + 8 * (-u_0[k*N*N+(j-1)*N+i+2] + u_0[k*N*N+(j+1)*N+i+2])))) + strx[i] * stry[j] * (1e0 / 144) * (la[k*N*N+(j-2)*N+i] * (u_0[k*N*N+(j-2)*N+i-2] - u_0[k*N*N+(j-2)*N+i+2] + 8 * (-u_0[k*N*N+(j-2)*N+i-1] + u_0[k*N*N+(j-2)*N+i+1])) - 8 * (la[k*N*N+(j-1)*N+i] * (u_0[k*N*N+(j-1)*N+i-2] - u_0[k*N*N+(j-1)*N+i+2] + 8 * (-u_0[k*N*N+(j-1)*N+i-1] + u_0[k*N*N+(j-1)*N+i+1]))) + 8 * (la[k*N*N+(j+1)*N+i] * (u_0[k*N*N+(j+1)*N+i-2] - u_0[k*N*N+(j+1)*N+i+2] + 8 * (-u_0[k*N*N+(j+1)*N+i-1] + u_0[k*N*N+(j+1)*N+i+1]))) - (la[k*N*N+(j+2)*N+i] * (u_0[k*N*N+(j+2)*N+i-2] - u_0[k*N*N+(j+2)*N+i+2] + 8 * (-u_0[k*N*N+(j+2)*N+i-1] + u_0[k*N*N+(j+2)*N+i+1])))) + stry[j] * strz[k] * (1e0 / 144) * (la[k*N*N+(j-2)*N+i] * (u_2[(k-2)*N*N+(j-2)*N+i] - u_2[(k+2)*N*N+(j-2)*N+i] + 8 * (-u_2[(k-1)*N*N+(j-2)*N+i] + u_2[(k+1)*N*N+(j-2)*N+i])) - 8 * (la[k*N*N+(j-1)*N+i] * (u_2[(k-2)*N*N+(j-1)*N+i] - u_2[(k+2)*N*N+(j-1)*N+i] + 8 * (-u_2[(k-1)*N*N+(j-1)*N+i] + u_2[(k+1)*N*N+(j-1)*N+i]))) + 8 * (la[k*N*N+(j+1)*N+i] * (u_2[(k-2)*N*N+(j+1)*N+i] - u_2[(k+2)*N*N+(j+1)*N+i] + 8 * (-u_2[(k-1)*N*N+(j+1)*N+i] + u_2[(k+1)*N*N+(j+1)*N+i]))) - (la[k*N*N+(j+2)*N+i] * (u_2[(k-2)*N*N+(j+2)*N+i] - u_2[(k+2)*N*N+(j+2)*N+i] + 8 * (-u_2[(k-1)*N*N+(j+2)*N+i] + u_2[(k+1)*N*N+(j+2)*N+i])))) + stry[j] * strz[k] * (1e0 / 144) * (mu[(k-2)*N*N+j*N+i] * (u_2[(k-2)*N*N+(j-2)*N+i] - u_2[(k-2)*N*N+(j+2)*N+i] + 8 * (-u_2[(k-2)*N*N+(j-1)*N+i] + u_2[(k-2)*N*N+(j+1)*N+i])) - 8 * (mu[(k-1)*N*N+j*N+i] * (u_2[(k-1)*N*N+(j-2)*N+i] - u_2[(k-1)*N*N+(j+2)*N+i] + 8 * (-u_2[(k-1)*N*N+(j-1)*N+i] + u_2[(k-1)*N*N+(j+1)*N+i]))) + 8 * (mu[(k+1)*N*N+j*N+i] * (u_2[(k+1)*N*N+(j-2)*N+i] - u_2[(k+1)*N*N+(j+2)*N+i] + 8 * (-u_2[(k+1)*N*N+(j-1)*N+i] + u_2[(k+1)*N*N+(j+1)*N+i]))) - (mu[(k+2)*N*N+j*N+i] * (u_2[(k+2)*N*N+(j-2)*N+i] - u_2[(k+2)*N*N+(j+2)*N+i] + 8 * (-u_2[(k+2)*N*N+(j-1)*N+i] + u_2[(k+2)*N*N+(j+1)*N+i])))); 
			r3 = r3 + strx[i] * strz[k] * (1e0 / 144) * (mu[k*N*N+j*N+i-2] * (u_0[(k-2)*N*N+j*N+i-2] - u_0[(k+2)*N*N+j*N+i-2] + 8 * (-u_0[(k-1)*N*N+j*N+i-2] + u_0[(k+1)*N*N+j*N+i-2])) - 8 * (mu[k*N*N+j*N+i-1] * (u_0[(k-2)*N*N+j*N+i-1] - u_0[(k+2)*N*N+j*N+i-1] + 8 * (-u_0[(k-1)*N*N+j*N+i-1] + u_0[(k+1)*N*N+j*N+i-1]))) + 8 * (mu[k*N*N+j*N+i+1] * (u_0[(k-2)*N*N+j*N+i+1] - u_0[(k+2)*N*N+j*N+i+1] + 8 * (-u_0[(k-1)*N*N+j*N+i+1] + u_0[(k+1)*N*N+j*N+i+1]))) - (mu[k*N*N+j*N+i+2] * (u_0[(k-2)*N*N+j*N+i+2] - u_0[(k+2)*N*N+j*N+i+2] + 8 * (-u_0[(k-1)*N*N+j*N+i+2] + u_0[(k+1)*N*N+j*N+i+2])))) + stry[j] * strz[k] * (1e0 / 144) * (mu[k*N*N+(j-2)*N+i] * (u_1[(k-2)*N*N+(j-2)*N+i] - u_1[(k+2)*N*N+(j-2)*N+i] + 8 * (-u_1[(k-1)*N*N+(j-2)*N+i] + u_1[(k+1)*N*N+(j-2)*N+i])) - 8 * (mu[k*N*N+(j-1)*N+i] * (u_1[(k-2)*N*N+(j-1)*N+i] - u_1[(k+2)*N*N+(j-1)*N+i] + 8 * (-u_1[(k-1)*N*N+(j-1)*N+i] + u_1[(k+1)*N*N+(j-1)*N+i]))) + 8 * (mu[k*N*N+(j+1)*N+i] * (u_1[(k-2)*N*N+(j+1)*N+i] - u_1[(k+2)*N*N+(j+1)*N+i] + 8 * (-u_1[(k-1)*N*N+(j+1)*N+i] + u_1[(k+1)*N*N+(j+1)*N+i]))) - (mu[k*N*N+(j+2)*N+i] * (u_1[(k-2)*N*N+(j+2)*N+i] - u_1[(k+2)*N*N+(j+2)*N+i] + 8 * (-u_1[(k-1)*N*N+(j+2)*N+i] + u_1[(k+1)*N*N+(j+2)*N+i])))) + strx[i] * strz[k] * (1e0 / 144) * (la[(k-2)*N*N+j*N+i] * (u_0[(k-2)*N*N+j*N+i-2] - u_0[(k-2)*N*N+j*N+i+2] + 8 * (-u_0[(k-2)*N*N+j*N+i-1] + u_0[(k-2)*N*N+j*N+i+1])) - 8 * (la[(k-1)*N*N+j*N+i] * (u_0[(k-1)*N*N+j*N+i-2] - u_0[(k-1)*N*N+j*N+i+2] + 8 * (-u_0[(k-1)*N*N+j*N+i-1] + u_0[(k-1)*N*N+j*N+i+1]))) + 8 * (la[(k+1)*N*N+j*N+i] * (u_0[(k+1)*N*N+j*N+i-2] - u_0[(k+1)*N*N+j*N+i+2] + 8 * (-u_0[(k+1)*N*N+j*N+i-1] + u_0[(k+1)*N*N+j*N+i+1]))) - (la[(k+2)*N*N+j*N+i] * (u_0[(k+2)*N*N+j*N+i-2] - u_0[(k+2)*N*N+j*N+i+2] + 8 * (-u_0[(k+2)*N*N+j*N+i-1] + u_0[(k+2)*N*N+j*N+i+1])))) + stry[j] * strz[k] * (1e0 / 144) * (la[(k-2)*N*N+j*N+i] * (u_1[(k-2)*N*N+(j-2)*N+i] - u_1[(k-2)*N*N+(j+2)*N+i] + 8 * (-u_1[(k-2)*N*N+(j-1)*N+i] + u_1[(k-2)*N*N+(j+1)*N+i])) - 8 * (la[(k-1)*N*N+j*N+i] * (u_1[(k-1)*N*N+(j-2)*N+i] - u_1[(k-1)*N*N+(j+2)*N+i] + 8 * (-u_1[(k-1)*N*N+(j-1)*N+i] + u_1[(k-1)*N*N+(j+1)*N+i]))) + 8 * (la[(k+1)*N*N+j*N+i] * (u_1[(k+1)*N*N+(j-2)*N+i] - u_1[(k+1)*N*N+(j+2)*N+i] + 8 * (-u_1[(k+1)*N*N+(j-1)*N+i] + u_1[(k+1)*N*N+(j+1)*N+i]))) - (la[(k+2)*N*N+j*N+i] * (u_1[(k+2)*N*N+(j-2)*N+i] - u_1[(k+2)*N*N+(j+2)*N+i] + 8 * (-u_1[(k+2)*N*N+(j-1)*N+i] + u_1[(k+2)*N*N+(j+1)*N+i]))));

			/* 3 * 3 = 9 flops */
			uacc_0[k*N*N+j*N+i] = a1 * uacc_0[k*N*N+j*N+i] + cof * r1;
			uacc_1[k*N*N+j*N+i] = a1 * uacc_1[k*N*N+j*N+i] + cof * r2;
			uacc_2[k*N*N+j*N+i] = a1 * uacc_2[k*N*N+j*N+i] + cof * r3;
		} 
}

extern "C" void host_code (float *h_uacc_0, float *h_uacc_1, float *h_uacc_2, float *h_u_0, float *h_u_1, float *h_u_2, float *h_mu, float *h_la, float *h_strx, float *h_stry, float *h_strz, int N) {
	float *uacc_0;
	hipMalloc (&uacc_0, sizeof(float)*N*N*N);
	check_error ("Failed to allocate device memory for uacc_0\n");
	hipMemcpy (uacc_0, h_uacc_0, sizeof(float)*N*N*N, hipMemcpyHostToDevice);
	float *uacc_1;
	hipMalloc (&uacc_1, sizeof(float)*N*N*N);
	check_error ("Failed to allocate device memory for uacc_1\n");
	hipMemcpy (uacc_1, h_uacc_1, sizeof(float)*N*N*N, hipMemcpyHostToDevice);
	float *uacc_2;
	hipMalloc (&uacc_2, sizeof(float)*N*N*N);
	check_error ("Failed to allocate device memory for uacc_2\n");
	hipMemcpy (uacc_2, h_uacc_2, sizeof(float)*N*N*N, hipMemcpyHostToDevice);
	float *u_0;
	hipMalloc (&u_0, sizeof(float)*N*N*N);
	check_error ("Failed to allocate device memory for u_0\n");
	hipMemcpy (u_0, h_u_0, sizeof(float)*N*N*N, hipMemcpyHostToDevice);
	float *u_1;
	hipMalloc (&u_1, sizeof(float)*N*N*N);
	check_error ("Failed to allocate device memory for u_1\n");
	hipMemcpy (u_1, h_u_1, sizeof(float)*N*N*N, hipMemcpyHostToDevice);
	float *u_2;
	hipMalloc (&u_2, sizeof(float)*N*N*N);
	check_error ("Failed to allocate device memory for u_2\n");
	hipMemcpy (u_2, h_u_2, sizeof(float)*N*N*N, hipMemcpyHostToDevice);
	float *mu;
	hipMalloc (&mu, sizeof(float)*N*N*N);
	check_error ("Failed to allocate device memory for mu\n");
	hipMemcpy (mu, h_mu, sizeof(float)*N*N*N, hipMemcpyHostToDevice);
	float *la;
	hipMalloc (&la, sizeof(float)*N*N*N);
	check_error ("Failed to allocate device memory for la\n");
	hipMemcpy (la, h_la, sizeof(float)*N*N*N, hipMemcpyHostToDevice);
	float *strx;
	hipMalloc (&strx, sizeof(float)*N);
	check_error ("Failed to allocate device memory for strx\n");
	hipMemcpy (strx, h_strx, sizeof(float)*N, hipMemcpyHostToDevice);
	float *stry;
	hipMalloc (&stry, sizeof(float)*N);
	check_error ("Failed to allocate device memory for stry\n");
	hipMemcpy (stry, h_stry, sizeof(float)*N, hipMemcpyHostToDevice);
	float *strz;
	hipMalloc (&strz, sizeof(float)*N);
	check_error ("Failed to allocate device memory for strz\n");
	hipMemcpy (strz, h_strz, sizeof(float)*N, hipMemcpyHostToDevice);

	dim3 blockconfig (32, 4, 2);
	dim3 gridconfig (ceil(N, blockconfig.x), ceil(N, blockconfig.y), ceil(N, blockconfig.z));

	sw4 <<<gridconfig, blockconfig>>> (uacc_0, uacc_1, uacc_2, u_0, u_1, u_2, mu, la, strx, stry, strz, N);

	hipMemcpy (h_uacc_0, uacc_0, sizeof(float)*N*N*N, hipMemcpyDeviceToHost);
	hipMemcpy (h_uacc_1, uacc_1, sizeof(float)*N*N*N, hipMemcpyDeviceToHost);
	hipMemcpy (h_uacc_2, uacc_2, sizeof(float)*N*N*N, hipMemcpyDeviceToHost);

	hipFree (uacc_0); 
	hipFree (uacc_1);
	hipFree (uacc_2);
	hipFree (u_0);
	hipFree (u_1);
	hipFree (u_2);
	hipFree (mu);
	hipFree (la);
	hipFree (strx);
	hipFree (stry);
	hipFree (strz);
}
