#include "hip/hip_runtime.h"
#include "stdio.h"

// extern __host__ __device__ int MAX(int a, int b) { return a > b ? a : b; }
// extern __host__ __device__ int MIN(int a, int b) { return a < b ? a : b; }
// extern __host__ __device__ int CEIL(int a, int b) { return ( (a) % (b) == 0 ? (a) / (b) :  ( (a) / (b) + 1 ) ); }

void Check_CUDA_Error(const char* message){
  hipError_t error = hipGetLastError();
  if( error != hipSuccess ){
    printf("CUDA-ERROR:%s, %s\n",message,hipGetErrorString(error) ); 
    exit(-1);
  }
}
