#include <stdio.h>
#include "hip/hip_runtime.h"
#define max(x,y)  ((x) > (y)? (x) : (y))
#define min(x,y)  ((x) < (y)? (x) : (y))
#define ceil(a,b) ((a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1)

void check_error (const char* message) {
  hipError_t error = hipGetLastError ();
  if (error != hipSuccess) {
    printf ("CUDA error : %s, %s\n", message, hipGetErrorString (error));
    exit(-1);
  }
}

__global__ void hypterm_1 (float * __restrict__ flux_0, float * __restrict__ flux_1, float * __restrict__ flux_2, float * __restrict__ flux_3, float * __restrict__ flux_4, float * __restrict__ cons_1, float * __restrict__ cons_2, float * __restrict__ cons_3, float * __restrict__ cons_4, float * __restrict__ q_1, float * __restrict__ q_2, float * __restrict__ q_3, float * __restrict__ q_4, float dxinv0, float dxinv1, float dxinv2, int L, int M, int N) {
  //Determing the block's indices
  int blockdim_i= (int)(blockDim.x);
  int i0 = (int)(blockIdx.x)*(blockdim_i);
  int i = max (i0, 0) + (int)(threadIdx.x);
  int blockdim_j= (int)(blockDim.y);
  int j0 = (int)(blockIdx.y)*(blockdim_j);
  int j = max (j0, 0) + (int)(threadIdx.y);
  int blockdim_k= (int)(blockDim.z);
  int k0 = (int)(blockIdx.z)*(blockdim_k);
  int k = max (k0, 0) + (int)(threadIdx.z);

  if (i>=4 & j>=4 & k>=4 & i<=N-5 & j<=N-5 & k<=N-5) {
  	flux_0[k*M*N+j*N+i] = -((0.8*(cons_1[k*M*N+j*N+i+1]-cons_1[k*M*N+j*N+i-1])-0.2*(cons_1[k*M*N+j*N+i+2]-cons_1[k*M*N+j*N+i-2])+0.038*(cons_1[k*M*N+j*N+i+3]-cons_1[k*M*N+j*N+i-3])-0.0035*(cons_1[k*M*N+j*N+i+4]-cons_1[k*M*N+j*N+i-4]))*dxinv0);
  	flux_1[k*M*N+j*N+i] = -((0.8*(cons_1[k*M*N+j*N+i+1]*q_1[k*M*N+j*N+i+1]-cons_1[k*M*N+j*N+i-1]*q_1[k*M*N+j*N+i-1]+(q_4[k*M*N+j*N+i+1]-q_4[k*M*N+j*N+i-1]))-0.2*(cons_1[k*M*N+j*N+i+2]*q_1[k*M*N+j*N+i+2]-cons_1[k*M*N+j*N+i-2]*q_1[k*M*N+j*N+i-2]+(q_4[k*M*N+j*N+i+2]-q_4[k*M*N+j*N+i-2]))+0.038*(cons_1[k*M*N+j*N+i+3]*q_1[k*M*N+j*N+i+3]-cons_1[k*M*N+j*N+i-3]*q_1[k*M*N+j*N+i-3]+(q_4[k*M*N+j*N+i+3]-q_4[k*M*N+j*N+i-3]))-0.0035*(cons_1[k*M*N+j*N+i+4]*q_1[k*M*N+j*N+i+4]-cons_1[k*M*N+j*N+i-4]*q_1[k*M*N+j*N+i-4]+(q_4[k*M*N+j*N+i+4]-q_4[k*M*N+j*N+i-4])))*dxinv0);
  	 flux_2[k*M*N+j*N+i] = -((0.8*(cons_2[k*M*N+j*N+i+1]*q_1[k*M*N+j*N+i+1]-cons_2[k*M*N+j*N+i-1]*q_1[k*M*N+j*N+i-1])-0.2*(cons_2[k*M*N+j*N+i+2]*q_1[k*M*N+j*N+i+2]-cons_2[k*M*N+j*N+i-2]*q_1[k*M*N+j*N+i-2])+0.038*(cons_2[k*M*N+j*N+i+3]*q_1[k*M*N+j*N+i+3]-cons_2[k*M*N+j*N+i-3]*q_1[k*M*N+j*N+i-3])-0.0035*(cons_2[k*M*N+j*N+i+4]*q_1[k*M*N+j*N+i+4]-cons_2[k*M*N+j*N+i-4]*q_1[k*M*N+j*N+i-4]))*dxinv0);
  	flux_3[k*M*N+j*N+i] = -((0.8*(cons_3[k*M*N+j*N+i+1]*q_1[k*M*N+j*N+i+1]-cons_3[k*M*N+j*N+i-1]*q_1[k*M*N+j*N+i-1])-0.2*(cons_3[k*M*N+j*N+i+2]*q_1[k*M*N+j*N+i+2]-cons_3[k*M*N+j*N+i-2]*q_1[k*M*N+j*N+i-2])+0.038*(cons_3[k*M*N+j*N+i+3]*q_1[k*M*N+j*N+i+3]-cons_3[k*M*N+j*N+i-3]*q_1[k*M*N+j*N+i-3])-0.0035*(cons_3[k*M*N+j*N+i+4]*q_1[k*M*N+j*N+i+4]-cons_3[k*M*N+j*N+i-4]*q_1[k*M*N+j*N+i-4]))*dxinv0);
  	flux_4[k*M*N+j*N+i] = -((0.8*(cons_4[k*M*N+j*N+i+1]*q_1[k*M*N+j*N+i+1]-cons_4[k*M*N+j*N+i-1]*q_1[k*M*N+j*N+i-1]+(q_4[k*M*N+j*N+i+1]*q_1[k*M*N+j*N+i+1]-q_4[k*M*N+j*N+i-1]*q_1[k*M*N+j*N+i-1]))-0.2*(cons_4[k*M*N+j*N+i+2]*q_1[k*M*N+j*N+i+2]-cons_4[k*M*N+j*N+i-2]*q_1[k*M*N+j*N+i-2]+(q_4[k*M*N+j*N+i+2]*q_1[k*M*N+j*N+i+2]-q_4[k*M*N+j*N+i-2]*q_1[k*M*N+j*N+i-2]))+0.038*(cons_4[k*M*N+j*N+i+3]*q_1[k*M*N+j*N+i+3]-cons_4[k*M*N+j*N+i-3]*q_1[k*M*N+j*N+i-3]+(q_4[k*M*N+j*N+i+3]*q_1[k*M*N+j*N+i+3]-q_4[k*M*N+j*N+i-3]*q_1[k*M*N+j*N+i-3]))-0.0035*(cons_4[k*M*N+j*N+i+4]*q_1[k*M*N+j*N+i+4]-cons_4[k*M*N+j*N+i-4]*q_1[k*M*N+j*N+i-4]+(q_4[k*M*N+j*N+i+4]*q_1[k*M*N+j*N+i+4]-q_4[k*M*N+j*N+i-4]*q_1[k*M*N+j*N+i-4])))*dxinv0);
  } 
}


__global__ void hypterm_2 (float * __restrict__ flux_0, float * __restrict__ flux_1, float * __restrict__ flux_2, float * __restrict__ flux_3, float * __restrict__ flux_4, float * __restrict__ cons_1, float * __restrict__ cons_2, float * __restrict__ cons_3, float * __restrict__ cons_4, float * __restrict__ q_1, float * __restrict__ q_2, float * __restrict__ q_3, float * __restrict__ q_4, float dxinv0, float dxinv1, float dxinv2, int L, int M, int N) {
  //Determing the block's indices
  int blockdim_i= (int)(blockDim.x);
  int i0 = (int)(blockIdx.x)*(blockdim_i);
  int i = max (i0, 0) + (int)(threadIdx.x);
  int blockdim_j= (int)(blockDim.y);
  int j0 = (int)(blockIdx.y)*(blockdim_j);
  int j = max (j0, 0) + (int)(threadIdx.y);
  int blockdim_k= (int)(blockDim.z);
  int k0 = (int)(blockIdx.z)*(blockdim_k);
  int k = max (k0, 0) + (int)(threadIdx.z);

  if (i>=4 & j>=4 & k>=4 & i<=N-5 & j<=N-5 & k<=N-5) {
  	flux_0[k*M*N+j*N+i] -= (0.8*(cons_2[k*M*N+(j+1)*N+i]-cons_2[k*M*N+(j-1)*N+i])-0.2*(cons_2[k*M*N+(j+2)*N+i]-cons_2[k*M*N+(j-2)*N+i])+0.038*(cons_2[k*M*N+(j+3)*N+i]-cons_2[k*M*N+(j-3)*N+i])-0.0035*(cons_2[k*M*N+(j+4)*N+i]-cons_2[k*M*N+(j-4)*N+i]))*dxinv1;
  	flux_1[k*M*N+j*N+i] -= (0.8*(cons_1[k*M*N+(j+1)*N+i]*q_2[k*M*N+(j+1)*N+i]-cons_1[k*M*N+(j-1)*N+i]*q_2[k*M*N+(j-1)*N+i])-0.2*(cons_1[k*M*N+(j+2)*N+i]*q_2[k*M*N+(j+2)*N+i]-cons_1[k*M*N+(j-2)*N+i]*q_2[k*M*N+(j-2)*N+i])+0.038*(cons_1[k*M*N+(j+3)*N+i]*q_2[k*M*N+(j+3)*N+i]-cons_1[k*M*N+(j-3)*N+i]*q_2[k*M*N+(j-3)*N+i])-0.0035*(cons_1[k*M*N+(j+4)*N+i]*q_2[k*M*N+(j+4)*N+i]-cons_1[k*M*N+(j-4)*N+i]*q_2[k*M*N+(j-4)*N+i]))*dxinv1;
  	flux_2[k*M*N+j*N+i] -= (0.8*(cons_2[k*M*N+(j+1)*N+i]*q_2[k*M*N+(j+1)*N+i]-cons_2[k*M*N+(j-1)*N+i]*q_2[k*M*N+(j-1)*N+i]+(q_4[k*M*N+(j+1)*N+i]-q_4[k*M*N+(j-1)*N+i]))-0.2*(cons_2[k*M*N+(j+2)*N+i]*q_2[k*M*N+(j+2)*N+i]-cons_2[k*M*N+(j-2)*N+i]*q_2[k*M*N+(j-2)*N+i]+(q_4[k*M*N+(j+2)*N+i]-q_4[k*M*N+(j-2)*N+i]))+0.038*(cons_2[k*M*N+(j+3)*N+i]*q_2[k*M*N+(j+3)*N+i]-cons_2[k*M*N+(j-3)*N+i]*q_2[k*M*N+(j-3)*N+i]+(q_4[k*M*N+(j+3)*N+i]-q_4[k*M*N+(j-3)*N+i]))-0.0035*(cons_2[k*M*N+(j+4)*N+i]*q_2[k*M*N+(j+4)*N+i]-cons_2[k*M*N+(j-4)*N+i]*q_2[k*M*N+(j-4)*N+i]+(q_4[k*M*N+(j+4)*N+i]-q_4[k*M*N+(j-4)*N+i])))*dxinv1;
  	flux_3[k*M*N+j*N+i] -= (0.8*(cons_3[k*M*N+(j+1)*N+i]*q_2[k*M*N+(j+1)*N+i]-cons_3[k*M*N+(j-1)*N+i]*q_2[k*M*N+(j-1)*N+i])-0.2*(cons_3[k*M*N+(j+2)*N+i]*q_2[k*M*N+(j+2)*N+i]-cons_3[k*M*N+(j-2)*N+i]*q_2[k*M*N+(j-2)*N+i])+0.038*(cons_3[k*M*N+(j+3)*N+i]*q_2[k*M*N+(j+3)*N+i]-cons_3[k*M*N+(j-3)*N+i]*q_2[k*M*N+(j-3)*N+i])-0.0035*(cons_3[k*M*N+(j+4)*N+i]*q_2[k*M*N+(j+4)*N+i]-cons_3[k*M*N+(j-4)*N+i]*q_2[k*M*N+(j-4)*N+i]))*dxinv1;
  	flux_4[k*M*N+j*N+i] -= (0.8*(cons_4[(k+1)*M*N+j*N+i]*q_3[(k+1)*M*N+j*N+i]-cons_4[(k-1)*M*N+j*N+i]*q_3[(k-1)*M*N+j*N+i]+(q_4[(k+1)*M*N+j*N+i]*q_3[(k+1)*M*N+j*N+i]-q_4[(k-1)*M*N+j*N+i]*q_3[(k-1)*M*N+j*N+i]))-0.2*(cons_4[(k+2)*M*N+j*N+i]*q_3[(k+2)*M*N+j*N+i]-cons_4[(k-2)*M*N+j*N+i]*q_3[(k-2)*M*N+j*N+i]+(q_4[(k+2)*M*N+j*N+i]*q_3[(k+2)*M*N+j*N+i]-q_4[(k-2)*M*N+j*N+i]*q_3[(k-2)*M*N+j*N+i]))+0.038*(cons_4[(k+3)*M*N+j*N+i]*q_3[(k+3)*M*N+j*N+i]-cons_4[(k-3)*M*N+j*N+i]*q_3[(k-3)*M*N+j*N+i]+(q_4[(k+3)*M*N+j*N+i]*q_3[(k+3)*M*N+j*N+i]-q_4[(k-3)*M*N+j*N+i]*q_3[(k-3)*M*N+j*N+i]))-0.0035*(cons_4[(k+4)*M*N+j*N+i]*q_3[(k+4)*M*N+j*N+i]-cons_4[(k-4)*M*N+j*N+i]*q_3[(k-4)*M*N+j*N+i]+(q_4[(k+4)*M*N+j*N+i]*q_3[(k+4)*M*N+j*N+i]-q_4[(k-4)*M*N+j*N+i]*q_3[(k-4)*M*N+j*N+i])))*dxinv2;
  } 
}


__global__ void hypterm_3 (float * __restrict__ flux_0, float * __restrict__ flux_1, float * __restrict__ flux_2, float * __restrict__ flux_3, float * __restrict__ flux_4, float * __restrict__ cons_1, float * __restrict__ cons_2, float * __restrict__ cons_3, float * __restrict__ cons_4, float * __restrict__ q_1, float * __restrict__ q_2, float * __restrict__ q_3, float * __restrict__ q_4, float dxinv0, float dxinv1, float dxinv2, int L, int M, int N) {
  //Determing the block's indices
  int blockdim_i= (int)(blockDim.x);
  int i0 = (int)(blockIdx.x)*(blockdim_i);
  int i = max (i0, 0) + (int)(threadIdx.x);
  int blockdim_j= (int)(blockDim.y);
  int j0 = (int)(blockIdx.y)*(blockdim_j);
  int j = max (j0, 0) + (int)(threadIdx.y);
  int blockdim_k= (int)(blockDim.z);
  int k0 = (int)(blockIdx.z)*(blockdim_k);
  int k = max (k0, 0) + (int)(threadIdx.z);

  if (i>=4 & j>=4 & k>=4 & i<=N-5 & j<=N-5 & k<=N-5) {
  	flux_0[k*M*N+j*N+i] -= (0.8*(cons_3[(k+1)*M*N+j*N+i]-cons_3[(k-1)*M*N+j*N+i])-0.2*(cons_3[(k+2)*M*N+j*N+i]-cons_3[(k-2)*M*N+j*N+i])+0.038*(cons_3[(k+3)*M*N+j*N+i]-cons_3[(k-3)*M*N+j*N+i])-0.0035*(cons_3[(k+4)*M*N+j*N+i]-cons_3[(k-4)*M*N+j*N+i]))*dxinv2;
  	flux_1[k*M*N+j*N+i] -= (0.8*(cons_1[(k+1)*M*N+j*N+i]*q_3[(k+1)*M*N+j*N+i]-cons_1[(k-1)*M*N+j*N+i]*q_3[(k-1)*M*N+j*N+i])-0.2*(cons_1[(k+2)*M*N+j*N+i]*q_3[(k+2)*M*N+j*N+i]-cons_1[(k-2)*M*N+j*N+i]*q_3[(k-2)*M*N+j*N+i])+0.038*(cons_1[(k+3)*M*N+j*N+i]*q_3[(k+3)*M*N+j*N+i]-cons_1[(k-3)*M*N+j*N+i]*q_3[(k-3)*M*N+j*N+i])-0.0035*(cons_1[(k+4)*M*N+j*N+i]*q_3[(k+4)*M*N+j*N+i]-cons_1[(k-4)*M*N+j*N+i]*q_3[(k-4)*M*N+j*N+i]))*dxinv2;
  	flux_2[k*M*N+j*N+i] -= (0.8*(cons_2[(k+1)*M*N+j*N+i]*q_3[(k+1)*M*N+j*N+i]-cons_2[(k-1)*M*N+j*N+i]*q_3[(k-1)*M*N+j*N+i])-0.2*(cons_2[(k+2)*M*N+j*N+i]*q_3[(k+2)*M*N+j*N+i]-cons_2[(k-2)*M*N+j*N+i]*q_3[(k-2)*M*N+j*N+i])+0.038*(cons_2[(k+3)*M*N+j*N+i]*q_3[(k+3)*M*N+j*N+i]-cons_2[(k-3)*M*N+j*N+i]*q_3[(k-3)*M*N+j*N+i])-0.0035*(cons_2[(k+4)*M*N+j*N+i]*q_3[(k+4)*M*N+j*N+i]-cons_2[(k-4)*M*N+j*N+i]*q_3[(k-4)*M*N+j*N+i]))*dxinv2;
  	flux_3[k*M*N+j*N+i] -= (0.8*(cons_3[(k+1)*M*N+j*N+i]*q_3[(k+1)*M*N+j*N+i]-cons_3[(k-1)*M*N+j*N+i]*q_3[(k-1)*M*N+j*N+i]+(q_4[(k+1)*M*N+j*N+i]-q_4[(k-1)*M*N+j*N+i]))-0.2*(cons_3[(k+2)*M*N+j*N+i]*q_3[(k+2)*M*N+j*N+i]-cons_3[(k-2)*M*N+j*N+i]*q_3[(k-2)*M*N+j*N+i]+(q_4[(k+2)*M*N+j*N+i]-q_4[(k-2)*M*N+j*N+i]))+0.038*(cons_3[(k+3)*M*N+j*N+i]*q_3[(k+3)*M*N+j*N+i]-cons_3[(k-3)*M*N+j*N+i]*q_3[(k-3)*M*N+j*N+i]+(q_4[(k+3)*M*N+j*N+i]-q_4[(k-3)*M*N+j*N+i]))-0.0035*(cons_3[(k+4)*M*N+j*N+i]*q_3[(k+4)*M*N+j*N+i]-cons_3[(k-4)*M*N+j*N+i]*q_3[(k-4)*M*N+j*N+i]+(q_4[(k+4)*M*N+j*N+i]-q_4[(k-4)*M*N+j*N+i])))*dxinv2;
  	flux_4[k*M*N+j*N+i] -= (0.8*(cons_4[k*M*N+(j+1)*N+i]*q_2[k*M*N+(j+1)*N+i]-cons_4[k*M*N+(j-1)*N+i]*q_2[k*M*N+(j-1)*N+i]+(q_4[k*M*N+(j+1)*N+i]*q_2[k*M*N+(j+1)*N+i]-q_4[k*M*N+(j-1)*N+i]*q_2[k*M*N+(j-1)*N+i]))-0.2*(cons_4[k*M*N+(j+2)*N+i]*q_2[k*M*N+(j+2)*N+i]-cons_4[k*M*N+(j-2)*N+i]*q_2[k*M*N+(j-2)*N+i]+(q_4[k*M*N+(j+2)*N+i]*q_2[k*M*N+(j+2)*N+i]-q_4[k*M*N+(j-2)*N+i]*q_2[k*M*N+(j-2)*N+i]))+0.038*(cons_4[k*M*N+(j+3)*N+i]*q_2[k*M*N+(j+3)*N+i]-cons_4[k*M*N+(j-3)*N+i]*q_2[k*M*N+(j-3)*N+i]+(q_4[k*M*N+(j+3)*N+i]*q_2[k*M*N+(j+3)*N+i]-q_4[k*M*N+(j-3)*N+i]*q_2[k*M*N+(j-3)*N+i]))-0.0035*(cons_4[k*M*N+(j+4)*N+i]*q_2[k*M*N+(j+4)*N+i]-cons_4[k*M*N+(j-4)*N+i]*q_2[k*M*N+(j-4)*N+i]+(q_4[k*M*N+(j+4)*N+i]*q_2[k*M*N+(j+4)*N+i]-q_4[k*M*N+(j-4)*N+i]*q_2[k*M*N+(j-4)*N+i])))*dxinv1;
  } 
}

extern "C" void host_code (float *h_flux_0, float *h_flux_1, float *h_flux_2, float *h_flux_3, float *h_flux_4, float *h_cons_1, float *h_cons_2, float *h_cons_3, float *h_cons_4, float *h_q_1, float *h_q_2, float *h_q_3, float *h_q_4, float dxinv0, float dxinv1, float dxinv2, int L, int M, int N) {
  float *flux_0;
  hipMalloc (&flux_0, sizeof(float)*L*M*N);
  check_error ("Failed to allocate device memory for flux_0\n");
  hipMemcpy (flux_0, h_flux_0, sizeof(float)*L*M*N, hipMemcpyHostToDevice);
  float *flux_1;
  hipMalloc (&flux_1, sizeof(float)*L*M*N);
  check_error ("Failed to allocate device memory for flux_1\n");
  hipMemcpy (flux_1, h_flux_1, sizeof(float)*L*M*N, hipMemcpyHostToDevice);
  float *flux_2;
  hipMalloc (&flux_2, sizeof(float)*L*M*N);
  check_error ("Failed to allocate device memory for flux_2\n");
  hipMemcpy (flux_2, h_flux_2, sizeof(float)*L*M*N, hipMemcpyHostToDevice);
  float *flux_3;
  hipMalloc (&flux_3, sizeof(float)*L*M*N);
  check_error ("Failed to allocate device memory for flux_3\n");
  hipMemcpy (flux_3, h_flux_3, sizeof(float)*L*M*N, hipMemcpyHostToDevice);
  float *flux_4;
  hipMalloc (&flux_4, sizeof(float)*L*M*N);
  check_error ("Failed to allocate device memory for flux_4\n");
  hipMemcpy (flux_4, h_flux_4, sizeof(float)*L*M*N, hipMemcpyHostToDevice);
  float *cons_1;
  hipMalloc (&cons_1, sizeof(float)*L*M*N);
  check_error ("Failed to allocate device memory for cons_1\n");
  hipMemcpy (cons_1, h_cons_1, sizeof(float)*L*M*N, hipMemcpyHostToDevice);
  float *cons_2;
  hipMalloc (&cons_2, sizeof(float)*L*M*N);
  check_error ("Failed to allocate device memory for cons_2\n");
  hipMemcpy (cons_2, h_cons_2, sizeof(float)*L*M*N, hipMemcpyHostToDevice);
  float *cons_3;
  hipMalloc (&cons_3, sizeof(float)*L*M*N);
  check_error ("Failed to allocate device memory for cons_3\n");
  hipMemcpy (cons_3, h_cons_3, sizeof(float)*L*M*N, hipMemcpyHostToDevice);
  float *cons_4;
  hipMalloc (&cons_4, sizeof(float)*L*M*N);
  check_error ("Failed to allocate device memory for cons_4\n");
  hipMemcpy (cons_4, h_cons_4, sizeof(float)*L*M*N, hipMemcpyHostToDevice);
  float *q_1;
  hipMalloc (&q_1, sizeof(float)*L*M*N);
  check_error ("Failed to allocate device memory for q_1\n");
  hipMemcpy (q_1, h_q_1, sizeof(float)*L*M*N, hipMemcpyHostToDevice);
  float *q_2;
  hipMalloc (&q_2, sizeof(float)*L*M*N);
  check_error ("Failed to allocate device memory for q_2\n");
  hipMemcpy (q_2, h_q_2, sizeof(float)*L*M*N, hipMemcpyHostToDevice);
  float *q_3;
  hipMalloc (&q_3, sizeof(float)*L*M*N);
  check_error ("Failed to allocate device memory for q_3\n");
  hipMemcpy (q_3, h_q_3, sizeof(float)*L*M*N, hipMemcpyHostToDevice);
  float *q_4;
  hipMalloc (&q_4, sizeof(float)*L*M*N);
  check_error ("Failed to allocate device memory for q_4\n");
  hipMemcpy (q_4, h_q_4, sizeof(float)*L*M*N, hipMemcpyHostToDevice);

  dim3 blockconfig (32, 4, 2);
  dim3 gridconfig (ceil(N, blockconfig.x), ceil(M, blockconfig.y), ceil(L, blockconfig.z));
  hypterm_1 <<<gridconfig, blockconfig>>> (flux_0, flux_1, flux_2, flux_3, flux_4, cons_1, cons_2, cons_3, cons_4, q_1, q_2, q_3, q_4, dxinv0, dxinv1, dxinv2, L, M, N);
  hypterm_2 <<<gridconfig, blockconfig>>> (flux_0, flux_1, flux_2, flux_3, flux_4, cons_1, cons_2, cons_3, cons_4, q_1, q_2, q_3, q_4, dxinv0, dxinv1, dxinv2, L, M, N);
  hypterm_3 <<<gridconfig, blockconfig>>> (flux_0, flux_1, flux_2, flux_3, flux_4, cons_1, cons_2, cons_3, cons_4, q_1, q_2, q_3, q_4, dxinv0, dxinv1, dxinv2, L, M, N);

  hipMemcpy (h_flux_0, flux_0, sizeof(float)*L*M*N, hipMemcpyDeviceToHost);
  hipMemcpy (h_flux_1, flux_1, sizeof(float)*L*M*N, hipMemcpyDeviceToHost);
  hipMemcpy (h_flux_3, flux_3, sizeof(float)*L*M*N, hipMemcpyDeviceToHost);
  hipMemcpy (h_flux_4, flux_4, sizeof(float)*L*M*N, hipMemcpyDeviceToHost);
  hipMemcpy (h_flux_2, flux_2, sizeof(float)*L*M*N, hipMemcpyDeviceToHost);

  hipFree(flux_0);
  hipFree(flux_1);
  hipFree(flux_2);
  hipFree(flux_3);
  hipFree(flux_4);
}
