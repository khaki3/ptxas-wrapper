#include <hip/hip_runtime.h>
#include <dlfcn.h>
#include <map>
#include <stdio.h>
#include <string>

using namespace std;

#ifdef __cplusplus
extern "C" {
#endif

#include "openacc_profiling.h"
#include "../timing.h"

static char* wrapper_funcname = 0;
static long wrapper_lineno = 0;

map<string, int> regcounts;

int kernelgen_enable_openacc_regcount(char* funcname, long lineno)
{
        wrapper_funcname = funcname;
        wrapper_lineno = lineno;
        return 0;
}

int kernelgen_disable_openacc_regcount()
{
        wrapper_funcname = 0;
        return 0;
}

void  __real_openacci_call(const char *file_name, int line_number, const char *function_name);

static const char* capture_kernel_launch = NULL;

void  __wrap_openacci_call(const char *file_name, int line_number, const char *function_name)
{
        if (__builtin_expect(wrapper_funcname != NULL, 1))
        {
                if (!strcmp(wrapper_funcname, file_name) &&
                        (wrapper_lineno == line_number))
                {
                        map<string, int>::iterator it = regcounts.find(function_name);
                        if (it == regcounts.end())
                        {
				// Capture and output regcount for the next launch.
				capture_kernel_launch = function_name;
			}
                        else
                        {
                                fprintf(stderr, "%s:%ld regcount = %d\n", wrapper_funcname,
                                        wrapper_lineno, it->second);
                        }
		}
	}

	__real_openacci_call(file_name, line_number, function_name);
}

bool timing = false;
struct timespec kernel_start, kernel_finish;

typedef hipError_t (*cuLaunchKernel_t)(
	hipFunction_t,
	unsigned int,
	unsigned int,
	unsigned int,
	unsigned int,
	unsigned int,
	unsigned int,
	unsigned int,
	hipStream_t,
	void **,
	void **);
static cuLaunchKernel_t cuLaunchKernel_ = NULL;

hipError_t hipModuleLaunchKernel(
	hipFunction_t f,
	unsigned int gridDimX,
	unsigned int gridDimY,
	unsigned int gridDimZ,
	unsigned int blockDimX,
	unsigned int blockDimY,
	unsigned int blockDimZ,
	unsigned int sharedMemBytes,
	hipStream_t hStream,
	void **kernelParams,
	void **extra)
{
	if (capture_kernel_launch)
	{
		int regcount = -1;
		hipError_t curesult = hipFuncGetAttribute(&regcount, HIP_FUNC_ATTRIBUTE_NUM_REGS, f);
		if (curesult != hipSuccess)
		{
			fprintf(stderr, "Failed to determine regcount for function %s\n", wrapper_funcname);
			exit(-1);
		}
		regcounts[capture_kernel_launch] = regcount;
		fprintf(stderr, "%s:%ld regcount = %d\n", wrapper_funcname,
			wrapper_lineno, regcount);
		capture_kernel_launch = NULL;
	}

	// Measure kernel time.
	get_time(&kernel_start);
	timing = true;

	return cuLaunchKernel_(f,
		gridDimX, gridDimY, gridDimZ,
		blockDimX, blockDimY, blockDimZ,
		sharedMemBytes, hStream, kernelParams, extra);
}

typedef hipError_t (*cuCtxSynchronize_t)();
static cuCtxSynchronize_t cuCtxSynchronize_ = NULL;

hipError_t hipCtxSynchronize()
{
	hipError_t result = cuCtxSynchronize_();

	if (timing)
	{
		get_time(&kernel_finish);
		fprintf(stderr, "%s:%ld time = %f\n", wrapper_funcname,
			wrapper_lineno, get_time_diff(&kernel_start, &kernel_finish));
		timing = false;
	}

	return result;
}

void* __libc_dlsym(void* handle, const char* symname) __THROW;

void* dlsym(void* handle, const char* symname) __THROW
{
	void* addr = __libc_dlsym(handle, symname);

	if (!strcmp(symname, "hipModuleLaunchKernel"))
	{
		cuLaunchKernel_ = (cuLaunchKernel_t)addr;
		return (void*)&hipModuleLaunchKernel;
	}
	if (!strcmp(symname, "hipCtxSynchronize"))
	{
		cuCtxSynchronize_ = (cuCtxSynchronize_t)addr;
		return (void*)&hipCtxSynchronize;
	}

	return addr;
}

#ifdef __cplusplus
}
#endif
