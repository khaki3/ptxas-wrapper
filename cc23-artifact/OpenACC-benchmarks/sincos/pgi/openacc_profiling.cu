//===----------------------------------------------------------------------===//
//
//     KernelGen -- A prototype of LLVM-based auto-parallelizing Fortran/C
//        compiler for NVIDIA GPUs, targeting numerical modeling code.
//
// This file is distributed under the University of Illinois Open Source
// License. See LICENSE.TXT for details.
//
//===----------------------------------------------------------------------===//

#include <hip/hip_runtime.h>
#include <map>
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <string.h>

using namespace std;

#ifdef __cplusplus
extern "C" {
#endif

static char* wrapper_funcname = 0;
static long wrapper_lineno = 0;

map<string, int> regcounts;

int kernelgen_enable_openacc_regcount(char* funcname, long lineno)
{
	wrapper_funcname = funcname;
	wrapper_lineno = lineno;
	return 0;
}

int kernelgen_disable_openacc_regcount()
{
	wrapper_funcname = 0;
	return 0;
}

struct uaccbinrec_t
{
	int binaryid;
	int fill;
	size_t binlen;
	char** binary;
};

struct uaccfuncrec_t
{
	int flags;
	int fill;
	long int lineno;
	char* functionname;
	// More args follow.
};

struct vinfo_t
{
	int magic;
	int flags;
	int pflags;
	int numfunctions;
	int numbinaries;
	int lock;
	void** handle;
	void** funchandle;
	uaccbinrec_t* bin;
	uaccfuncrec_t* func;
};

void __real___pgi_uacc_cuda_launch(vinfo_t* vinfo, int funcnum, void* argptr,
	long int* sargs, long int async, int dindex);

void __wrap___pgi_uacc_cuda_launch(vinfo_t* vinfo, int funcnum, void* argptr,
        long int* sargs, long int async, int dindex)
{
	if (__builtin_expect(wrapper_funcname != NULL, 1))
	{
		char* funcname = vinfo->func->functionname;
		long int lineno = vinfo->func->lineno;
		if (!strcmp(wrapper_funcname, funcname) &&
			(wrapper_lineno == lineno))
		{
			map<string, int>::iterator it = regcounts.find(funcname);
			if (it == regcounts.end())
			{
				// Get the register count for the underlying image.
				hipModule_t module;
				hipError_t curesult = hipModuleLoadData(&module, (char*)vinfo->bin->binary);
				if (curesult != hipSuccess)
				{
					fprintf(stderr, "Failed to load module from handle %p\n", vinfo->bin->binary);
					exit(-1);				
				}
				hipFunction_t func;
				curesult = hipModuleGetFunction(&func, module, funcname);
				if (curesult != hipSuccess)
				{
					fprintf(stderr, "Failed to load function %s from module handle %p\n",
						funcname, vinfo->bin->binary);
					exit(-1);
				}
				int regcount = -1;
				curesult = hipFuncGetAttribute(&regcount, HIP_FUNC_ATTRIBUTE_NUM_REGS, func);
				if (curesult != hipSuccess)
				{
					fprintf(stderr, "Failed to determine regcount for function %s\n", funcname);
					exit(-1);
				}
				regcounts[funcname] = regcount;
				curesult = hipModuleUnload(module);
				if (curesult != hipSuccess)
				{
					fprintf(stderr, "Failed to unload module from handle %p\n", vinfo->bin->binary);
					exit(-1);				
				}
				fprintf(stderr, "%s:%ld regcount = %d\n", wrapper_funcname,
					lineno, regcount);
			}
			else
			{
				fprintf(stderr, "%s:%ld regcount = %d\n", wrapper_funcname,
					lineno, it->second);
			}
		}
	}

	__real___pgi_uacc_cuda_launch(vinfo, funcnum, argptr, sargs, async, dindex);
}

#ifdef __cplusplus
}
#endif

